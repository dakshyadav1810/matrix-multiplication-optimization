
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

const int SIZE = 100;
const int BLOCK_SIZE = 16;

// CUDA kernel to perform matrix multiplication with shared memory
__global__ void matrix_multiply_cuda(int *A, int *B, int *C, int size)
{
    __shared__ int As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int Bs[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int val = 0;
    for (int m = 0; m < size / BLOCK_SIZE; ++m)
    {
        // Load data into shared memory
        As[threadIdx.y][threadIdx.x] = A[row * size + (m * BLOCK_SIZE + threadIdx.x)];
        Bs[threadIdx.y][threadIdx.x] = B[(m * BLOCK_SIZE + threadIdx.y) * size + col];
        __syncthreads();

        // Compute the matrix multiplication within the block
        for (int k = 0; k < BLOCK_SIZE; ++k)
        {
            val += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        }
        __syncthreads();
    }

    C[row * size + col] = val;
}

int main()
{
    int A[SIZE][SIZE];
    int B[SIZE][SIZE];
    int C[SIZE][SIZE];

    // Initialize matrices with 1s
    for (int i = 0; i < SIZE; ++i)
    {
        for (int j = 0; j < SIZE; ++j)
        {
            A[i][j] = 1;
            B[i][j] = 1;
            C[i][j] = 0; // Initialize result matrix to 0
        }
    }

    // Declare device pointers
    int *d_A, *d_B, *d_C;

    // Allocate device memory
    hipMalloc(&d_A, SIZE * SIZE * sizeof(int));
    hipMalloc(&d_B, SIZE * SIZE * sizeof(int));
    hipMalloc(&d_C, SIZE * SIZE * sizeof(int));

    // Copy input matrices from host to device
    hipMemcpy(d_A, A, SIZE * SIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, SIZE * SIZE * sizeof(int), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE); // 16x16 threads per block
    dim3 numBlocks((SIZE + threadsPerBlock.x - 1) / threadsPerBlock.x, (SIZE + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Start timing
    auto start = std::chrono::steady_clock::now();

    // Launch CUDA kernel
    matrix_multiply_cuda<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, SIZE);

    // End timing
    auto end = std::chrono::steady_clock::now();
    auto diff = end - start;
    std::cout << std::chrono::duration<double, std::milli>(diff).count() << std::endl;

    // Copy result matrix from device to host
    hipMemcpy(C, d_C, SIZE * SIZE * sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
